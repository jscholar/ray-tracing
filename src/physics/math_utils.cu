#include "hip/hip_runtime.h"
#include "physics/math_utils.cuh"

#include <limits>
#include <memory>
#include <random>

#include "physics/scattering.cuh"
#include "physics/vec3.cuh"

double math_utils::random_double() {
    static std::uniform_real_distribution<double> distribution(0.0, 1.0);
    static std::mt19937 generator;
    return distribution(generator);
}

vec3 math_utils::random_vector(double magnitude) {
	double x = (math_utils::random_double() * 2) - 1;
	double y = (math_utils::random_double() * 2) - 1;
	double z = (math_utils::random_double() * 2) - 1;

	vec3 new_random_vector = vec3(x, y, z);

	if (new_random_vector.length() <= 1) {
		return unit_vector(new_random_vector) * magnitude;
	}
	else {
		return math_utils::random_vector(magnitude);
	}
}

// TODO: Negative inputs
vec3 math_utils::random_vector(double min_magnitude, double max_magnitude) {
	if (min_magnitude > max_magnitude) {
		std::cout << "Received (min: " << min_magnitude << ")" << " and (max: " << max_magnitude << "), inferring the opposite";
		return random_vector(max_magnitude, min_magnitude);
	}

	// Generate a uniformly random magnitude.
	const double delta = std::pow(random_double(), 1 / 3) * max_magnitude - min_magnitude;
	const double magnitude =  std::abs(min_magnitude + delta);

	return random_vector(magnitude);
}
