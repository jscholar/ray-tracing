#include "physics/ray_tracer.cuh"

#include "physics/vec3.cuh"
#include "physics/ray.cuh"
#include "physics/scattering.cuh"
#include "objects/hittable.cuh"
#include "objects/hittable_object.cuh"
#include "objects/material.cuh"

#include <memory>
#include <algorithm>
#include <hip/hip_runtime.h>

color ray_tracer::ray_color(const ray& r, const hittable& hittable_entity, int bounce) {
	if (bounce <= 5) {
		std::optional<hit_record> possible_hit = hittable_entity.find_hit(r, 0.001, 100.0);

		if (possible_hit.has_value()) {
			hit_record hrec = possible_hit.value();
			point3 p = r.at(hrec.t);

			// Get scatter 
			std::shared_ptr<material> mat = hrec.object_hit->p_material;
			ray s = ray(p, mat->get_bounce_direction(r.dir, hrec.normal));


			// ray s = ray(p, hrec.object_hit->*p_material->get_bounce_direction(hrec.object_hit->outward_normal_at(p)));

			// Get attenuation
			color attenuating_color = mat->get_reflectiveness();

			color new_color = ray_color(s, hittable_entity, bounce + 1);

			// Prevents white speckles
			new_color[0] = std::clamp(new_color[0], 0.0, 1.0);
			new_color[1] = std::clamp(new_color[1], 0.0, 1.0);
			new_color[2] = std::clamp(new_color[2], 0.0, 1.0);
			
			return attenuating_color * new_color;
			// return hrec.object_hit->p_material->get_reflectiveness() * ray_color(s, hittable_entity, bounce + 1);
		}
	}

	// The sky/background color
	vec3 unit_direction = unit_vector(r.direction());
	auto t = 0.5 * (unit_direction.y() + 1.0);
	return (1.0 - t) * color(0.08, 0.08, 0.25) + t * color(0.08, 0.08, 0.15);
}

vec3 ray_tracer::ray_scatter(vec3& incoming_direction, const material& material_hit, vec3& surface_normal) {
	return vec3();
}