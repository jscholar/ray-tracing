#include "io/color.h"
#include "physics/vec3.cuh"
#include "physics/ray.cuh"
#include "physics/scattering.cuh"
#include "physics/math_utils.cuh"
#include "physics/ray_tracer.cuh"
#include "objects/hittable.cuh"
#include "objects/sphere.cuh"
#include "objects/hit_list.cuh"
#include "objects/material.cuh"
#include "objects/sphere.cuh"

#include <fstream>
#include <string>
#include <vector>
#include <optional>
#include <memory>
#include <hip/driver_types.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>


std::string file_name{ "example-image.ppm" };

int main() {

	std::ofstream image_file;
	image_file.open(file_name);

	// Image
	const auto aspect_ratio = 16.0 / 9.0;
	const int image_width = 1920;
	const int image_height = static_cast<int>(image_width / aspect_ratio);

	// Camera
	auto num_samples = 10;
	auto viewport_height = 2.0;
	auto viewport_width = aspect_ratio * viewport_height;
	auto focal_length = 2.0;

	auto origin = point3(0, 0, 0);
	auto horizontal = vec3(viewport_width, 0, 0);
	auto vertical = vec3(0, viewport_height, 0);
	auto lower_left_corner = origin - horizontal / 2 - vertical / 2 - vec3(0, 0, focal_length);

	// Materials
	auto mat_stone = std::make_shared<stone>();
	auto mat_grass = std::make_shared<grass>();
	auto mat_ocean = std::make_shared<ocean>();
	auto mat_fluorescent = std::make_shared<fluorescent_white>();
	auto mat_gold = std::make_shared<gold>();
	auto mat_metal = std::make_shared<metal>();

	std::shared_ptr<sphere> sphere2 = std::make_shared<sphere>(sphere(point3(0, 1, -8.5), 1.5, color(0.0, 1.0, 0.0), mat_gold));
	std::shared_ptr<sphere> sphere3 = std::make_shared<sphere>(sphere(point3(-2.5, 0.5, -10), 1.0, color(0.0, 1.0, 0.0), mat_metal));
	std::shared_ptr<sphere> sphere5 = std::make_shared<sphere>(sphere(point3(0.25, -0.25, -6.5), 0.25, color(0.0, 1.0, 0.0), mat_metal));

	std::shared_ptr<sphere> hill1 = std::make_shared<sphere>(sphere(point3(3.5, -1.25, -4.5), 1.3, color(0.0, 1.0, 0.0), mat_grass));

	std::shared_ptr<sphere> mountain1 = std::make_shared<sphere>(sphere(point3(-25, -15, -30), 30, color(0.0, 1.0, 0.0), mat_stone));

	std::shared_ptr<sphere> moon = std::make_shared<sphere>(sphere(point3(25, 15, -75), 15.0, color(0.0, 1.0, 0.0), mat_fluorescent));
	std::shared_ptr<sphere> moon2 = std::make_shared<sphere>(sphere(point3(55, 23, -75), 5.0, color(0.0, 1.0, 0.0), mat_metal));
	std::shared_ptr<sphere> earth = std::make_shared<sphere>(sphere(point3(0, -200.5, -1), 200.0, color(0.0, 1.0, 0.0), mat_ocean));

	std::vector<std::shared_ptr<hittable>> sphere_vec = { sphere2, sphere3, sphere5, hill1, mountain1, earth, moon, moon2 };
	hit_list spheres = hit_list(sphere_vec);

	// Render
	image_file << "P3\n" << image_width << " " << image_height << "\n255\n";

	for (int j = image_height - 1; j >= 0; --j) {
		std::cerr << "\rScanlines remaining: " << j << ' ' << std::flush;
		for (int i = 0; i < image_width; ++i) {

			color stacked_color = color(0, 0, 0);
			for (int sample_number = 0; sample_number < num_samples; ++sample_number) {
				auto u = (i + math_utils::random_double()) / (image_width - 1);
				auto v = (j + math_utils::random_double()) / (image_height - 1);

				ray r(origin, lower_left_corner + u * horizontal + v * vertical - origin);
				color pixel_color = ray_tracer::ray_color(r, spheres, 0);

				stacked_color += pixel_color;
			}

			stacked_color /= num_samples;
			write_color(image_file, stacked_color);
		}
	}

	std::cerr << "\nDone.\n";
}
