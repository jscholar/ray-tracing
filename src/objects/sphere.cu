#include "hip/hip_runtime.h"
﻿#include "objects/sphere.cuh"

#include "physics/vec3.cuh"
#include "physics/ray.cuh"
#include "physics/scattering.cuh"
#include "objects/material.cuh"
#include "objects/hittable.cuh"
#include "objects/hittable_object.cuh"

#include <optional>
#include <math.h>

sphere::sphere(const point3& center, double radius, const color& surface_color, const std::shared_ptr<material> p_material)
	: center{ center }, radius{ radius }, surfaceColor{ surface_color }, hittable_object{ p_material } {};

std::optional<hit_record> sphere::find_hit(const ray& ray, double t_min, double t_max) const {
	point3 A = ray.orig;
	vec3 b = ray.dir;
	point3 C = center;

	// t^2b.b
	double q_a = dot(b, b);

	// 2tb.(A−C)
	double q_b = 2 * dot(b, A - C);

	// (A−C).(A−C)−r^2
	double q_c = dot(A - C, A - C) - (radius * radius);

	// Check direction of the ray
	double discriminant = (q_b * q_b) - (4 * q_a * q_c);
	if (discriminant < 0) {
		return std::nullopt;
	}

	double root = (-q_b - sqrt(discriminant)) / (2 * q_a);

	// Check range of the ray
	if (root > t_max || root < t_min) {
		root = (-q_b + sqrt(discriminant)) / (2 * q_a);
		if (root > t_max || root < t_min) {
			return std::nullopt;
		}
	}

	hit_record hrec;
	hrec.t = root;
	hrec.normal = outward_normal_at(ray.at(hrec.t));
	if (dot(ray.dir, hrec.normal) > 0) {
		hrec.normal *= -1;
	}
	hrec.object_hit = shared_from_this();

	return hrec;
}

vec3 sphere::outward_normal_at(const point3& point) const {
	return unit_vector(point - center);
}
